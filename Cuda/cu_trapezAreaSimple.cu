#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <time.h>


__host__ __device__ double f(double x){
  return x*x;
}

__device__ double atomicAddDouble(double* address, double val) {
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
                    __double_as_longlong(val + __longlong_as_double(assumed)));
  } while (assumed != old);
  return __longlong_as_double(old);
}

__global__ void integrate(double a, double b, int n, double h, double *d_integral) {
  int i;
  double x, sum = 0.0;
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  
  for (i = idx + 1; i < n; i += stride) {
    x = a + i * h;
    sum += f(x);
  }
  
  sum *= 2.0;
  
  atomicAddDouble(d_integral, sum);
}

double trapezoidalMethod(double a, double b, int n, double h) {
    int i;
    double x, sum = 0;

    for (i = 1; i < n; i++) {
        x = a + i * h;
        sum += f(x);
    }

    return (h / 2) * (f(a) + f(b) + 2 * sum);
}

int main() {
    int n = 1000000, blockSize, numBlocks;;
    double a = 50, b = 1000000, h, integral;
    double *d_integral;
    double *dev_integral;

    h = fabs(b - a) / n;
    
    clock_t start_s = clock();

    integral = trapezoidalMethod(a, b, n, h);

    clock_t end_s = clock();

    blockSize = 256;
    numBlocks = (n + blockSize - 1) / blockSize;
  
    d_integral = (double*)malloc(sizeof(double));
    hipMalloc((void**)&dev_integral, sizeof(double));
  
    *d_integral = 0.0;
    hipMemcpy(dev_integral, d_integral, sizeof(double), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    integrate<<<numBlocks, blockSize>>>(a, b, n, h, dev_integral);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipMemcpy(d_integral, dev_integral, sizeof(double), hipMemcpyDeviceToHost);
  
    integral = (h / 2) * (f(a) + f(b) + *d_integral);

    float parallel_time = 0;
    int numThreads = numBlocks * blockSize;
    double sequential_time = (double)(end_s - start_s) / CLOCKS_PER_SEC;


    hipEventElapsedTime(&parallel_time, start, stop);
    double speedup = sequential_time / parallel_time;
    double scalability = speedup / numThreads;

    printf("\nEl valor de la integral es: %lf\n", integral);
    printf("Tiempo de ejecución en paralelo: %lf segundos\n", parallel_time/1000);
    printf("Tiempo de ejecución secuencial: %lf segundos\n", sequential_time);
    printf("Speedup: %lf\n", speedup);
    printf("Escalabilidad: %lf\n", scalability);

    hipFree(d_integral);
    hipFree(dev_integral);

    return 0;
}
